
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<29;
  float *x, *y;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  add<<<1, 256>>>(N, x, y);


  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(x);
  hipFree(y);
  
  return 0;
}

